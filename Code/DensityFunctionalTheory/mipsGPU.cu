#include "hip/hip_runtime.h"
typedef struct {
    float l;
    float epsilon;
    float sigma;
    float v0;
} potential_param;

__device__ float potential(float3 r1, float3 r2, potential_param p) {
    float dx = r1.x - r2.x;
    float dy = r1.y - r2.y;
    float atemp = dx > 0 ? 1. : -1.;
    if(dx*dx > 0.25*p.l*p.l) {
        dx = dx - atemp*p.l;
    }
    float btemp = dy > 0 ? 1. : -1.;
    if(dy*dy > 0.25*p.l*p.l) {
        dy = dy - btemp*p.l;
    }

    float rSQR = SQR(dx) + SQR(dy);

    //float fac1 = (sigma / sqrt(rSQR));
    float fac2 = SQR(p.sigma)/rSQR;
    //else expf = 0.0;

    float st;
    if (rSQR > 0.69 * SQR(p.sigma))
    { //cut off
        float f1 = ((4. * p.epsilon * p.v0)) * (4 * fac2 * fac2 - 2 * fac2); //long range potential
        st = f1 * (dx * cos(r1.z) - dx * cos(r2.z) + dy * sin(r1.z) - dy * sin(r2.z));
    }
    else
    {
        st = 0.0;
    }

    return st;
}


// __global__ void calc_ints(float *den, float *res, int tot) {
// 	//particle positions in dev_unc_pos
// 	//dev cell list is the list of all particles with their cell
// 	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

// 	if(global_index < tot) {
// 		dev_cell_list[global_index] = nbox*floorf(nbox*(dev_unc_pos[global_index].x)/lcell) +  floorf(nbox*(dev_unc_pos[global_index].y)/lcell);
// 		//dev_cell_list[global_index].y =
//     }
    
// }

template <class Q>
void print_device_array(Q *array, int n) {
Q *temparray = new Q [n];
hipMemcpy(temparray,array,n*sizeof(Q),hipMemcpyDeviceToHost);
for(int i = 0 ; i < n ; i++) {
	cout << temparray[i] <<  ",";
}
cout << endl;
delete temparray;

}


__global__ void assign_array(float *my_array, float len, int tot) {
   int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if(global_index < tot) {
         float dl = len/(float)(tot);
         my_array[global_index] = global_index*dl;
     }
}

__global__  void assign_pos(float3 *pos, float *xpos, float *qpos, int Nr, int Nq) {
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if(global_index < Nr*Nr*Nq) {
        float3 a;
        int q = global_index/SQR(Nr);
        int temp = global_index - q*SQR(Nr);
        int i = temp/Nr;
        int j = temp % Nr;

        a.x = xpos[i];
        a.y = xpos[j];
        a.z = qpos[q];
        pos[global_index] = a;
    }
}

__global__ void assign_den(float* den, float val, int n) {
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if(global_index < n) {
        den[global_index] = val;
    }
}

__global__ void convo(float *den, 
    float3 *points_in_space,
    int con,
    float *res,
    potential_param p,
    float volume_element, int tot) 
    {
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;

    if(global_index < tot) {
        //int q = global_index % Nr

        res[global_index] =  volume_element* den[global_index] * potential(points_in_space[con],points_in_space[global_index],p);
    }
}


__global__ void calcint(float *den, float3 *points_in_space, float *res, potential_param p, float volume_element, int tot) {
    // int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    // if(global_index < tot) {
    //     float *newres = new float [tot];

    //     convo(den,points_in_space,global_index,newres,p,volume_element,tot);

    //     thrust::device_ptr<float> t_int(newres);
    //     res[global_index] = thrust::reduce(t_int,t_int+tot);
        

        
    // }

}