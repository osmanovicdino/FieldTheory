#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdarg.h>
#include <vector>
#include <algorithm>
#include <stdexcept>
#include <limits>
#include <cmath>
#include <complex>
#include <sstream>
#include <string>
#include <iomanip>
#include <sys/ioctl.h> 
#include <fcntl.h>
#include <time.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <random>
#if defined(_OPENMP)
#include <omp.h>
#else
typedef int omp_int_t;
inline omp_int_t omp_get_thread_num() { return 0;}
inline omp_int_t omp_get_max_threads() { return 1;}
inline omp_int_t omp_get_num_threads() { return 1; }
#endif

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/unique.h>
#include <thrust/device_delete.h>
#include <thrust/system_error.h>
#include <thrust/system/cuda/error.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include "DataStructures/basic.h"
#include "DataStructures/vector1.h"
#include "DataStructures/matrix2.h"
#include "DataStructures/matrix2.cpp"


// #include "BrownianGel.cpp"
// #include "BrownianGel2.cpp"
// #include "LangevinGel.cpp"
// #include "LangevinGelFixed.cpp"

// #include "NCGasR.h"
#include "DensityFunctionalTheory/mipsGPU.cu"



using namespace std;

int main(int argc, char** argv) {
srand (time(NULL));


    int Nr = 200;
    int Nq = 40;

	float3 *d_pos;

	int size2 = Nr*Nr*Nq*sizeof(float3);


    hipMalloc((void**)&d_pos,size2);

    float *d_xpos;
    float *d_qpos;

    int size3 = Nr*sizeof(float);
    int size4 = Nq*sizeof(float);

    hipMalloc((void**)&d_xpos,size3);
    hipMalloc((void**)&d_qpos,size4);
    
    int n = 1000;
    float l = 40.0;

    float volume_element = l*l*2.*pi/(float(Nr)*float(Nr)*float(Nq));

    assign_array<<<Nr,1>>>(d_xpos, l, Nr);
    assign_array<<<Nr,1>>>(d_qpos, 2*pi, Nq);

    assign_pos<<<Nr*Nr,Nq>>>(d_pos, d_xpos, d_qpos, Nr,Nq);

    potential_param p;
    p.l = l;
    p.epsilon = 1.0;
    p.sigma = 1.0;
    p.v0 = 1.0;

    float *d_den;
    hipMalloc((void**)&d_den,Nr*Nr*Nq*sizeof(float));

    float val = (float)n/(l*l*2*pi);
    assign_den<<<Nr*Nr,Nq>>>(d_den,val,Nr*Nr*Nq);



    float *d_int;
    hipMalloc((void**)&d_int,Nr*Nr*Nq*sizeof(float));
    for(int i = 0 ; i < Nr*Nr*Nq ; i++) {
    convo<<<Nr*Nr,Nq>>>(d_den, 
    d_pos,
    i,
    d_int,
    p,
    volume_element, Nr*Nr*Nq); 

    thrust::device_ptr<float> t_int(d_int);
   float tot = thrust::reduce(t_int,t_int+Nr*Nr*Nq);

    cout << i << " " << tot <<endl;
    }

return 0;

}